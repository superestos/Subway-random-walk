#include "hip/hip_runtime.h"
#include "../shared/globals.hpp"
#include "../shared/timer.hpp"
#include "../shared/argument_parsing.cuh"
#include "../shared/graph.cuh"
#include "../shared/subgraph.cuh"
#include "../shared/partitioner.cuh"
#include "../shared/subgraph_generator.cuh"
#include "../shared/gpu_error_check.cuh"
#include "../shared/gpu_kernels.cuh"
#include "../shared/subway_utilities.hpp"
#include "../shared/stopwatch.h"

int main(int argc, char** argv)
{
	Stopwatch copyTimer;
	Stopwatch computeTimer;

	hipFree(0);

	ArgumentParser arguments(argc, argv, true, false);
	
	Timer timer;
	timer.Start();
	
	Graph<OutEdge> graph(arguments.input, false);
	graph.ReadGraph();
	
	float readtime = timer.Finish();
	cout << "Graph Reading finished in " << readtime/1000 << " (s).\n";
	
	for(unsigned int i=0; i<graph.num_nodes; i++)
	{
		graph.value[i] = DIST_INFINITY;
		graph.label1[i] = false;
		graph.label2[i] = false;
	}
	graph.value[arguments.sourceNode] = 0;
	graph.label1[arguments.sourceNode] = false;
	graph.label2[arguments.sourceNode] = true;


	gpuErrorcheck(hipMemcpy(graph.d_outDegree, graph.outDegree, graph.num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(graph.d_value, graph.value, graph.num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(graph.d_label1, graph.label1, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(graph.d_label2, graph.label2, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice));
	
	Subgraph<OutEdge> subgraph(graph.num_nodes, graph.num_edges);
	
	SubgraphGenerator<OutEdge> subgen(graph);
	copyTimer.start();
	subgen.generate(graph, subgraph);
	copyTimer.stop();

	Partitioner<OutEdge> partitioner;
	
	timer.Start();
	
	uint itr = 0;

	unsigned long totalActiveNodes = 0;
		
	while (subgraph.numActiveNodes>0)
	{
		itr++;
		
		partitioner.partition(subgraph, subgraph.numActiveNodes);

		cout << "num active nodes: " << subgraph.numActiveNodes << "\n";
		totalActiveNodes += subgraph.numActiveNodes;

		// a super iteration
		for(int i=0; i<partitioner.numPartitions; i++)
		{
			copyTimer.start();
			hipDeviceSynchronize();
			gpuErrorcheck(hipMemcpy(subgraph.d_activeEdgeList, subgraph.activeEdgeList + partitioner.fromEdge[i], (partitioner.partitionEdgeSize[i]) * sizeof(OutEdge), hipMemcpyHostToDevice));
			hipDeviceSynchronize();
			copyTimer.stop();

			computeTimer.start();
			moveUpLabels<<< partitioner.partitionNodeSize[i]/512 + 1 , 512 >>>(subgraph.d_activeNodes, graph.d_label1, graph.d_label2, partitioner.partitionNodeSize[i], partitioner.fromNode[i]);

			bfs_kernel<<< partitioner.partitionNodeSize[i]/512 + 1 , 512 >>>(partitioner.partitionNodeSize[i],
													partitioner.fromNode[i],
													partitioner.fromEdge[i],
													subgraph.d_activeNodes,
													subgraph.d_activeNodesPointer,
													subgraph.d_activeEdgeList,
													graph.d_outDegree,
													graph.d_value, 
													//d_finished,
													graph.d_label1,
													graph.d_label2);

			hipDeviceSynchronize();
			computeTimer.stop();
			gpuErrorcheck( hipPeekAtLastError() );	
		}
		
		copyTimer.start();
		subgen.generate(graph, subgraph);
		copyTimer.stop();
	}
	
	float runtime = timer.Finish();
	cout << "Processing finished in " << runtime/1000 << " (s).\n";
	
	cout << "Number of iterations = " << itr << endl;

	cout << "compute time: " << computeTimer.total() << " ns copy time: " << copyTimer.total() << " ns\n";

	cout << "total active nodes: " << totalActiveNodes << "\n";
	
	gpuErrorcheck(hipMemcpy(graph.value, graph.d_value, graph.num_nodes*sizeof(uint), hipMemcpyDeviceToHost));
	
	utilities::PrintResults(graph.value, min(30, graph.num_nodes));
			
	if(arguments.hasOutput)
		utilities::SaveResults(arguments.output, graph.value, graph.num_nodes);
}

