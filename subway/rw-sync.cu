#include "hip/hip_runtime.h"
#include "../shared/globals.hpp"
#include "../shared/timer.hpp"
#include "../shared/argument_parsing.cuh"
#include "../shared/graph.cuh"
#include "../shared/subgraph.cuh"
#include "../shared/partitioner.cuh"
#include "../shared/subgraph_generator.cuh"
#include "../shared/gpu_error_check.cuh"
#include "../shared/gpu_kernels.cuh"
#include "../shared/subway_utilities.hpp"
#include "../shared/test.cuh"
#include "../shared/test.cu"
#include "../shared/stopwatch.h"



int main(int argc, char** argv)
{	
	Stopwatch copyTimer;
	Stopwatch computeTimer;
	Stopwatch generateTimer;

	hipFree(0);

	ArgumentParser arguments(argc, argv, true, false);
	
	Timer timer;
	timer.Start();
	
	GraphPR<OutEdge> graph(arguments.input, true);
	graph.ReadGraph();
	
	float readtime = timer.Finish();
	cout << "Graph Reading finished in " << readtime/1000 << " (s).\n";

	int *numWalker1 = new int[graph.num_nodes];
	int *d_numWalker1, *d_numWalker2;

	hipMalloc(&d_numWalker1, sizeof(int) * graph.num_nodes);
	hipMalloc(&d_numWalker2, sizeof(int) * graph.num_nodes);

	hiprandState *randStates;
	hipMalloc(&randStates, sizeof(hiprandState) * 512);
	init_rand<<<1, 512>>>(randStates, 512);

	std::random_device rd;
	std::mt19937 rng{rd()}; 
	std::uniform_int_distribution<int> uniform(0, graph.num_nodes - 1);
	
	for(unsigned int i=0; i<graph.num_nodes; i++)
	{
		graph.value[i] = 0;
		numWalker1[i] = 0;
	}

	u_int32_t numWalker = graph.num_nodes * 2;
	for(u_int32_t i = 0; i < numWalker; i++) {
		numWalker1[uniform(rng)]++;
	}

	gpuErrorcheck(hipMemcpy(graph.d_outDegree, graph.outDegree, graph.num_nodes * sizeof(u_int64_t), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(graph.d_value, graph.value, graph.num_nodes * sizeof(float), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_numWalker1, numWalker1, graph.num_nodes * sizeof(int), hipMemcpyHostToDevice));
	
	Subgraph<OutEdge> subgraph(graph.num_nodes, graph.num_edges);
	
	SubgraphGenerator<OutEdge> subgen(graph);
	
	generateTimer.start();
	subgen.generate(graph, subgraph, d_numWalker1);
	generateTimer.stop();

	cout << "generate subgraph" << endl;

	Partitioner<OutEdge> partitioner;
	
	timer.Start();
	
	uint gItr = 0;

	unsigned long totalActiveNodes = 0;

	u_int32_t *visit_count;
	hipMalloc(&visit_count, sizeof(u_int32_t) * graph.num_nodes);
	u_int32_t *h_visit_count = new u_int32_t[graph.num_nodes];
		
	for (; gItr < 10; gItr++)
	{
		
		partitioner.partition(subgraph, subgraph.numActiveNodes);

		cout << "num active nodes: " << subgraph.numActiveNodes << "\n";

		u_int64_t activeEdges = 0;
		for(int i=0; i<partitioner.numPartitions; i++)
		{
			activeEdges += partitioner.partitionEdgeSize[i];
		}

		cout << "num active edges: " << activeEdges << "\n";
		
		totalActiveNodes += subgraph.numActiveNodes;

		// a super iteration
		for(int i=0; i<partitioner.numPartitions; i++)
		{
			hipDeviceSynchronize();
			copyTimer.start();
			gpuErrorcheck(hipMemcpy(subgraph.d_activeEdgeList, subgraph.activeEdgeList + partitioner.fromEdge[i], (partitioner.partitionEdgeSize[i]) * sizeof(OutEdge), hipMemcpyHostToDevice));
			hipDeviceSynchronize();
			copyTimer.stop();
			
			computeTimer.start();
			rw_kernel<<< partitioner.partitionNodeSize[i]/512 + 1 , 512 >>>(graph.num_nodes,
												partitioner.partitionNodeSize[i],
												partitioner.fromNode[i],
												partitioner.fromEdge[i],
												subgraph.d_activeNodes,
												subgraph.d_activeNodesPointer,
												subgraph.d_activeEdgeList,
												graph.d_outDegree,
												graph.d_value,
												d_numWalker1,
												d_numWalker2,
												randStates);		

			//moveWalkers<<< partitioner.partitionNodeSize[i]/512 + 1 , 512 >>>(subgraph.d_activeNodes, d_numWalker1, d_numWalker2, graph.d_value, partitioner.partitionNodeSize[i], partitioner.fromNode[i]);

			hipDeviceSynchronize();
			computeTimer.stop();
			gpuErrorcheck( hipPeekAtLastError() );	
	
		}

		//computeTimer.stop();
		moveWalkers_pr<<<graph.num_nodes/512 + 1, 512>>>(graph.num_nodes, d_numWalker1, d_numWalker2, visit_count);
		//hipDeviceSynchronize();
		//computeTimer.stop();
		
		generateTimer.start();
		subgen.generate(graph, subgraph, d_numWalker1);
		generateTimer.stop();
	}	
	
	float runtime = timer.Finish();
	cout << "Processing finished in " << runtime/1000 << " (s).\n";
	
	cout << "Number of iterations = " << gItr << endl;

	cout << "compute time: " << computeTimer.total() << " ns copy time: " << copyTimer.total() << " ns\n";
	cout << "generate subgraph time: " << generateTimer.total() << " ns\n";

	cout << "total active nodes: " << totalActiveNodes << "\n";
	
	gpuErrorcheck(hipMemcpy(h_visit_count, visit_count, graph.num_nodes*sizeof(u_int32_t), hipMemcpyDeviceToHost));

	unsigned long sum = 0;
	for (unsigned i = 0; i < graph.num_nodes; i++) {
		sum += h_visit_count[i];
	}
	cout << "sum: " << sum << endl;
	
	utilities::PrintResults(graph.value, min(30, graph.num_nodes));

			
	if(arguments.hasOutput)
		utilities::SaveResults(arguments.output, graph.value, graph.num_nodes);
}

